#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiply(const float *A, const float *B, float *C, int numElements)
{
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float counter = 0;

    if (ROW < numElements && COL < numElements) 
    {
        for (int i = 0; i < numElements; i++) 
        {
            counter += A[ROW * numElements + i] * B[i * numElements + COL];
        }
    }

    C[ROW * numElements + COL] = counter;
}

int main(void)
{
    int numElements = 1000; 
    int threadsPerBlock = 512; 
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    size_t size = numElements * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    int matrixOrder = 4;

    for (int i = 0; i < matrixOrder; i++)
    {
        for (int j = 0; j < matrixOrder; j++)
        {
            h_A[(i * matrixOrder) + j] = rand()/(float)RAND_MAX;
            h_B[(i * matrixOrder) + j] = rand()/(float)RAND_MAX;
        }
    }

    float *d_A = NULL;  hipMalloc((void **)&d_A, size);
    float *d_B = NULL;  hipMalloc((void **)&d_B, size);
    float *d_C = NULL;  hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t start,stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    double memXFers=5*4*COLUMNS*ROWS;
    memXFers/=1024*1024*1024;

    printf("GPU: %f ms bandwidth %g GB/s",ms, memXFers/(ms/1000.0));
    printf("\n CPU : %g ms bandwidth %g GB/s",mtime, memXFers/(mtime/1000.0));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}